#define LIMIT -999

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define BLOCK_SIZE 16
#define ROWS       64 
#define COLS       64
#define PENALTY    10


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);


__device__ __host__ int maximum( int a, int b, int c){
  int k;
  if( a <= b )
    k = b;
  else 
    k = a;

  if( k <=c )
    return(c);
  else
    return(k);
}

__global__ void
needle_cuda_shared_1( int* reference,
                      int* matrix_cuda, 
                      int* matrix_cuda_out, 
                      int cols,
                      int penalty,
                      int i,
                      int block_width) 
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx;
  int b_index_y = i - 1 - bx;

  int index    = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( cols + 1 );
  int index_n  = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( 1 );
  int index_w  = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols );
  int index_nw = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x;

  __shared__  int temp[BLOCK_SIZE+1][BLOCK_SIZE+1];
  __shared__  int ref[BLOCK_SIZE][BLOCK_SIZE];

  for ( int ty = 0 ; ty < BLOCK_SIZE ; ty++) {
    ref[ty][tx] = reference[index + cols * ty];
  }
  __syncthreads();

  if (tx == 0) {
    temp[tx][0] = matrix_cuda[index_nw];
  }

  temp[tx + 1][0] = matrix_cuda[index_w + cols * tx];
  __syncthreads();

  temp[0][tx + 1] = matrix_cuda[index_n];
  __syncthreads();

  for( int m = 0 ; m < BLOCK_SIZE ; m++){
    if ( tx <= m ){
      int t_index_x =  tx + 1;
      int t_index_y =  m - tx + 1;
      temp[t_index_y][t_index_x] = maximum( temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
                                            temp[t_index_y][t_index_x-1]  - penalty, 
                                            temp[t_index_y-1][t_index_x]  - penalty);
    }
    __syncthreads();
  }

  for( int m = BLOCK_SIZE - 2 ; m >=0 ; m--){ 
    if ( tx <= m){
      int t_index_x =  tx + BLOCK_SIZE - m ;
      int t_index_y =  BLOCK_SIZE - tx;

      temp[t_index_y][t_index_x] = maximum( temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
                                            temp[t_index_y][t_index_x-1]  - penalty, 
                                            temp[t_index_y-1][t_index_x]  - penalty);
    }
    __syncthreads();
  }

  for ( int ty = 0 ; ty < BLOCK_SIZE ; ty++) {
    matrix_cuda[index + ty * cols] = temp[ty+1][tx+1];
  }
}


__global__ void
needle_cuda_shared_2( int* reference,
		      int* matrix_cuda, 
		      int* matrix_cuda_out, 
		      int cols,
		      int penalty,
		      int i,
		      int block_width) 
{

  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx + block_width - i  ;
  int b_index_y = block_width - bx -1;

  int index   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( cols + 1 );
  int index_n   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( 1 );
  int index_w   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols );
  int index_nw =  cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x;

  __shared__  int temp[BLOCK_SIZE+1][BLOCK_SIZE+1];
  __shared__  int ref[BLOCK_SIZE][BLOCK_SIZE];

  for ( int ty = 0 ; ty < BLOCK_SIZE ; ty++) {
    ref[ty][tx] = reference[index + cols * ty];
  }
  __syncthreads();

  if (tx == 0) {
    temp[tx][0] = matrix_cuda[index_nw];
  }

  temp[tx + 1][0] = matrix_cuda[index_w + cols * tx];
  __syncthreads();

  temp[0][tx + 1] = matrix_cuda[index_n];
  __syncthreads();
  

  for( int m = 0 ; m < BLOCK_SIZE ; m++) {
    if ( tx <= m ){
      int t_index_x =  tx + 1;
      int t_index_y =  m - tx + 1;

      temp[t_index_y][t_index_x] = maximum( temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
                                            temp[t_index_y][t_index_x-1]  - penalty, 
                                            temp[t_index_y-1][t_index_x]  - penalty);	    
    }
    __syncthreads();
  }

  for( int m = BLOCK_SIZE - 2 ; m >=0 ; m--) { 
    if ( tx <= m){
      int t_index_x =  tx + BLOCK_SIZE - m ;
      int t_index_y =  BLOCK_SIZE - tx;

      temp[t_index_y][t_index_x] = maximum( temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
                                            temp[t_index_y][t_index_x-1]  - penalty, 
                                            temp[t_index_y-1][t_index_x]  - penalty);
    }
    __syncthreads();
  }

  for ( int ty = 0 ; ty < BLOCK_SIZE ; ty++) {
    matrix_cuda[index + ty * cols] = temp[ty+1][tx+1];
  }
}

__global__ void
needle_cuda_plain_1( int* reference,
                      int* matrix_cuda, 
                      int* matrix_cuda_out, 
                      int cols,
                      int penalty,
                      int i,
                      int block_width) 
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx;
  int b_index_y = i - 1 - bx;

  int index    = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols + 1 );

  for( int m = 0 ; m < BLOCK_SIZE ; m++){
    if ( tx <= m ){
      int t_index_x = tx;
      int t_index_y = m - tx;
      int idx = index + t_index_y * cols + t_index_x;
      matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
                                  matrix_cuda[idx - 1]    - penalty, 
                                  matrix_cuda[idx - cols] - penalty);
    }
  }

  for( int m = BLOCK_SIZE - 2 ; m >=0 ; m--){ 
    if ( tx <= m){
      int t_index_x =  tx + BLOCK_SIZE - m -1;
      int t_index_y =  BLOCK_SIZE - tx - 1;
      int idx = index + t_index_y * cols + t_index_x;
      matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
                                  matrix_cuda[idx - 1]    - penalty, 
                                  matrix_cuda[idx - cols] - penalty);
    }
  }
}

__global__ void
needle_cuda_plain_2( int* reference,
		      int* matrix_cuda, 
		      int* matrix_cuda_out, 
		      int cols,
		      int penalty,
		      int i,
		      int block_width) 
{

  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx + block_width - i;
  int b_index_y = block_width - bx -1;

  int index    = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols + 1 );

  for( int m = 0 ; m < BLOCK_SIZE ; m++) {
    if ( tx <= m ){
      int t_index_x = tx;
      int t_index_y = m - tx;
      int idx = index + t_index_y * cols + t_index_x;
      matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
                                  matrix_cuda[idx - 1]    - penalty, 
                                  matrix_cuda[idx - cols] - penalty);
    }
  }

  for( int m = BLOCK_SIZE - 2 ; m >=0 ; m--) { 
    if ( tx <= m){
      int t_index_x =  tx + BLOCK_SIZE - m -1;
      int t_index_y =  BLOCK_SIZE - tx - 1;
      int idx = index + t_index_y * cols + t_index_x;
      matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
                                  matrix_cuda[idx - 1]    - penalty, 
                                  matrix_cuda[idx - cols] - penalty);
    }
  }
}

int blosum62[24][24] = {
{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
};

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
  runTest( argc, argv);

  return( 0);
}

void usage(int argc, char **argv)
{
  fprintf(stderr, "Usage: %s <max_rows/max_cols> <penalty> \n", argv[0]);
  fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
  fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
  exit(1);
}

void runTest( int argc, char** argv) 
{
  int max_rows, max_cols, penalty;
  int *input_itemsets, *output_itemsets, *referrence;
  int *matrix_cuda, *matrix_cuda_out, *referrence_cuda;
  int size;
	
  // the lengths of the two sequences should be able to divided by 16.
  // And at current stage  max_rows needs to equal max_cols
  max_rows = ROWS;
  max_cols = COLS;
  penalty = PENALTY;
	
  if(max_rows%BLOCK_SIZE!=0 || max_cols%BLOCK_SIZE!=0) {
    fprintf(stderr,"The dimension values must be a multiple of 16\n");
    exit(1);
  }
	
  max_rows = max_rows + 1;
  max_cols = max_cols + 1;
  referrence = (int *)malloc( max_rows * max_cols * sizeof(int));
  input_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int));
  output_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int));
	
  if (!input_itemsets) {
    fprintf(stderr, "error: can not allocate memory");
  }
  srand ( 7 );
	
  for (int i = 0 ; i < max_cols; i++){
    for (int j = 0 ; j < max_rows; j++){
      input_itemsets[i*max_cols+j] = 0;
    }
  }
	
  printf("Start Needleman-Wunsch\n");
	
  for( int i=1; i< max_rows ; i++){    //please define your own sequence. 
    input_itemsets[i*max_cols] = rand() % 10 + 1;
  }
  for( int j=1; j< max_cols ; j++){    //please define your own sequence.
    input_itemsets[j] = rand() % 10 + 1;
  }

  for (int i = 1 ; i < max_cols; i++){
    for (int j = 1 ; j < max_rows; j++){
      referrence[i*max_cols+j] = blosum62[input_itemsets[i*max_cols]][input_itemsets[j]];
    }
  }

  for( int i = 1; i< max_rows ; i++) {
    input_itemsets[i*max_cols] = -i * penalty;
  }

  for( int j = 1; j< max_cols ; j++) {
    input_itemsets[j] = -j * penalty;
  }

  size = max_cols * max_rows;
  hipMalloc((void**)& referrence_cuda, sizeof(int)*size);
  hipMalloc((void**)& matrix_cuda, sizeof(int)*size);
  hipMalloc((void**)& matrix_cuda_out, sizeof(int)*size);
	
  hipMemcpy(referrence_cuda, referrence, sizeof(int) * size, hipMemcpyHostToDevice);
  hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size, hipMemcpyHostToDevice);

  dim3 dimGrid;
  dim3 dimBlock(BLOCK_SIZE, 1);
  int block_width = ( max_cols - 1 )/BLOCK_SIZE;

#ifdef PLAIN  /* No shared memory optimization */
  printf("Processing top-left matrix\n");
  //process top-left matrix
  for( int i = 1 ; i <= block_width ; i++){
    dimGrid.x = i;
    dimGrid.y = 1;
    cudaFuncSetCacheConfig("needle_cuda_plain_1", cudaFuncCachePreferL1);
    needle_cuda_plain_1<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda, matrix_cuda_out ,max_cols, penalty, i, block_width); 
  }

  printf("Processing bottom-right matrix\n");
  //process bottom-right matrix
  for( int i = block_width - 1  ; i >= 1 ; i--){
    dimGrid.x = i;
    dimGrid.y = 1;
    cudaFuncSetCacheConfig("needle_cuda_plain_2", cudaFuncCachePreferL1);
    needle_cuda_plain_2<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda, matrix_cuda_out ,max_cols, penalty, i, block_width); 
  }
#else 
  printf("Processing top-left matrix\n");
  //process top-left matrix
  for( int i = 1 ; i <= block_width ; i++){
    dimGrid.x = i;
    dimGrid.y = 1;
    needle_cuda_shared_1<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda, matrix_cuda_out ,max_cols, penalty, i, block_width); 
  }

  printf("Processing bottom-right matrix\n");
  //process bottom-right matrix
  for( int i = block_width - 1  ; i >= 1 ; i--){
    dimGrid.x = i;
    dimGrid.y = 1;
    needle_cuda_shared_2<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda, matrix_cuda_out ,max_cols, penalty, i, block_width); 
  }
#endif

  hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size, hipMemcpyDeviceToHost);
	
#ifdef OUTPUT
/*
  printf("print traceback value GPU:\n");
    
  for (int i = max_rows - 2,  j = max_rows - 2; i>=0, j>=0;){
    int nw, n, w, traceback;

    if ( i == max_rows - 2 && j == max_rows - 2 ) {
      printf("%d ", output_itemsets[ i * max_cols + j]); //print the first element
    }
        
    if ( i == 0 && j == 0) {
      break;
    }

    if ( i > 0 && j > 0 ){
      nw = output_itemsets[(i - 1) * max_cols + j - 1];
      w  = output_itemsets[ i * max_cols + j - 1 ];
      n  = output_itemsets[(i - 1) * max_cols + j];
    } else if ( i == 0 ){
      nw = n = LIMIT;
      w  = output_itemsets[ i * max_cols + j - 1 ];
    } else if ( j == 0 ){
      nw = w = LIMIT;
      n  = output_itemsets[(i - 1) * max_cols + j];
    } else{ }

    traceback = maximum(nw, w, n);
		
    printf("%d ", traceback);

    if(traceback == nw ) {i--; j--; continue;}
    else if(traceback == w ) {j--; continue;}
    else if(traceback == n ) {i--; continue;}
    else {}
  }
  printf("\n");
*/

  int i, j;
  for( i = 0; i < max_rows; i++) {
    for( j = 0; j < max_cols; j++) {
      printf("%d ", output_itemsets[i*max_cols+j]);
    }
    printf("\n");
  }
#endif

  hipFree(referrence_cuda);
  hipFree(matrix_cuda);
  hipFree(matrix_cuda_out);

}
