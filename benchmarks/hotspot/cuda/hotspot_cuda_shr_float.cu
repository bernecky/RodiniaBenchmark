
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

//#define USEFILE
/* For reading and writing file output */

#define BLOCK_SIZE 16
#define STR_SIZE 256

#define ITER 5000

#ifndef SIZE
#define SIZE 1024
#endif

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	     3000000.0f
/* required precision in degrees	*/
#define PRECISION    0.001f
#define SPEC_HEAT_SI 1750000.0f
#define K_SI         100
/* capacitance fitting factor	*/
#define FACTOR_CHIP  0.5f

/* chip parameters	*/
float t_chip = 0.0005f;
float chip_height = 0.016f;
float chip_width = 0.016f;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0f;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

static inline
double range_random(int min, int max)
{ 
  long int n;
  do 
  { 
    n=random(); 
  } while (n==RAND_MAX);
  
  return((((n%RAND_MAX)/(double)RAND_MAX) * (max-min+1) + min));
}

static
void generateinput(float * powerar, float * tempar, int grid_rows, int grid_cols)
{ // It is assumed that both *ar's have been allocated
  int i,j;

  srandom( 7);

  for(i = 0; i < grid_rows; i++)
  {
    for(j = 0; j < grid_cols; j++)
    {
      powerar[i*grid_cols+j] = range_random(4, 706)/1000000.0;
      tempar[i*grid_cols+j] = range_random(322980566, 343964157)/1000000.0;
    }
  }
}

void fatal(char *s)
{
  fprintf(stderr, "error: %s\n", s);
}

void writeoutput(float *vect, int grid_rows, int grid_cols, char *file)
{
/*
  int i,j, index=0;
  FILE *fp;
  char str[STR_SIZE];

  if( (fp = fopen(file, "w" )) == 0 ) {
    printf( "The file was not opened\n" );
  }

  for (i=0; i < grid_rows; i++) { 
    for (j=0; j < grid_cols; j++) {
      sprintf(str, "%g\n", vect[i*grid_cols+j]);
      fputs(str,fp);
      index++;
    }
  }
  fclose(fp);	
*/
#ifdef OUTPUT
  int i,j;
  for (i=0; i < grid_rows; i++) { 
    for (j=0; j < grid_cols; j++) {
      printf("%f\n", vect[i*grid_cols+j]);
    }
  }
#else
  printf("%f\n", vect[0]);
#endif
}


void readinput(float *vect, int grid_rows, int grid_cols, char *file)
{
  int i,j;
  FILE *fp;
  char str[STR_SIZE];
  float val;

  if( (fp  = fopen(file, "r" )) ==0 )
    printf( "The file was not opened\n" );

  for (i=0; i <= grid_rows-1; i++) { 
    for (j=0; j <= grid_cols-1; j++) {
      fgets(str, STR_SIZE, fp);
      if (feof(fp))
        fatal("not enough lines in file");
	//if ((sscanf(str, "%d%f", &index, &val) != 2) || (index != ((i-1)*(grid_cols-2)+j-1)))
      if ((sscanf(str, "%f", &val) != 1))
        fatal("invalid file format");
      vect[i*grid_cols+j] = val;
    }
  }
  fclose(fp);	
}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(int iteration,      //number of iteration
                               float *power,       //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               int grid_cols,      //Col of grid
                               int grid_rows,      //Row of grid
                               int border_cols,    // border offset 
			       int border_rows,    // border offset
                               //float Cap,          
                               float Rx_1, 
                               float Ry_1, 
                               float Rz_1, 
                               float step_div_Cap) 
{	
  __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

  float amb_temp = 80.0f;
        
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx=threadIdx.x;
  int ty=threadIdx.y;
	
/*	
  Rx_1=1/Rx;
  Ry_1=1/Ry;
  Rz_1=1/Rz;
*/
	
  // each block finally computes result for a small block
  // after N iterations. 
  // it is the non-overlapping small blocks that cover 
  // all the input data

  // calculate the small block size
  int small_block_rows = BLOCK_SIZE-iteration*2; //EXPAND_RATE
  int small_block_cols = BLOCK_SIZE-iteration*2; //EXPAND_RATE

  // calculate the boundary for the block according to 
  // the boundary of its small block
  int blkY = small_block_rows*by-border_rows;
  int blkX = small_block_cols*bx-border_cols;
  int blkYmax = blkY+BLOCK_SIZE-1;
  int blkXmax = blkX+BLOCK_SIZE-1;

  // calculate the global thread coordination
  int yidx = blkY+ty;
  int xidx = blkX+tx;

  // load data if it is within the valid input range
  int loadYidx=yidx, loadXidx=xidx;
  int index = grid_rows*loadYidx+loadXidx;
       
  if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
    temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
    power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory
  }
  __syncthreads();

  // effective range within this block that falls within 
  // the valid range of the input data
  // used to rule out computation outside the boundary.
  int validYmin = (blkY < 0) ? -blkY : 0;
  int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
  int validXmin = (blkX < 0) ? -blkX : 0;
  int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

  int N = ty-1;
  int S = ty+1;
  int W = tx-1;
  int E = tx+1;
        
  N = (N < validYmin) ? validYmin : N;
  S = (S > validYmax) ? validYmax : S;
  W = (W < validXmin) ? validXmin : W;
  E = (E > validXmax) ? validXmax : E;

  bool computed;
  for (int i=0; i<iteration ; i++) { 
    computed = false;
    if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  
        IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  
        IN_RANGE(tx, validXmin, validXmax) && 
        IN_RANGE(ty, validYmin, validYmax) ) {
      computed = true;
      temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
	       	         (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0f*temp_on_cuda[ty][tx]) * Ry_1 + 
                         (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0f*temp_on_cuda[ty][tx]) * Rx_1 + 
		         (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
    }
    __syncthreads();
    if(i==iteration-1) {
      break;
    }

    if(computed) {	 //Assign the computation range
      temp_on_cuda[ty][tx]= temp_t[ty][tx];
    }
    __syncthreads();
  }

  // update the global memory
  // after the last iteration, only threads coordinated within the 
  // small block perform the calculation and switch on ``computed''
  if (computed) {
    temp_dst[index]= temp_t[ty][tx];		
  }
}

/*
   compute N time steps
*/
int compute_tran_temp(float *MatrixPower,float *MatrixTemp[2], int col, int row, 
		      int total_iterations, int num_iterations, int blockCols, 
                      int blockRows, int borderCols, int borderRows) 
{
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(blockCols, blockRows);  
	
  float grid_height = chip_height / row;
  float grid_width = chip_width / col;

  float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
  float Rx = grid_width / (2.0f * K_SI * t_chip * grid_height);
  float Ry = grid_height / (2.0f * K_SI * t_chip * grid_width);
  float Rz = t_chip / (K_SI * grid_height * grid_width);
  float Rx_1 = 1.0f/Rx; 
  float Ry_1 = 1.0f/Ry;
  float Rz_1 = 1.0f/Rz;

  float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
  float step = PRECISION / max_slope;
  float t;

  float step_div_Cap = step/Cap;

  int src = 1, dst = 0;
  int count = 0;

  struct timeval tv1, tv2;
  gettimeofday( &tv1, NULL);
	
  for (t = 0; t < total_iterations; t+=num_iterations) {
    int temp = src;
    src = dst;
    dst = temp;
    calculate_temp<<<dimGrid, dimBlock>>>( MIN(num_iterations, total_iterations-t), 
                                           MatrixPower,MatrixTemp[src],MatrixTemp[dst],
                                           col,row,borderCols, borderRows, Rx_1,Ry_1,Rz_1,
                                           step_div_Cap);

    //cudaThreadSynchronize();
    count++;
  }

  gettimeofday( &tv2, NULL);
  double runtime = ((tv2.tv_sec+ tv2.tv_usec/1000000.0)-(tv1.tv_sec+ tv1.tv_usec/1000000.0));
  printf("Runtime(seconds): %f\n", runtime);
  printf("kernel has been executed for %d times\n", count);

  return dst;
}

void usage(int argc, char **argv)
{
#ifdef USEFILE
  fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
  fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
  fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
  fprintf(stderr, "\t<sim_time>   - number of iterations\n");
  fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
  fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
  fprintf(stderr, "\t<output_file> - name of the output file\n");
#else
  fprintf(stderr, "Usage: %s\n", argv[0]);
  fprintf(stderr, "\tAll key parameters are encoded as C-Macros, see source for details.\n");
#endif
  exit(1);
}

int main(int argc, char** argv)
{
  run(argc,argv);

  return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
  int size;
  int grid_rows,grid_cols;
  float *FilesavingTemp,*FilesavingPower,*MatrixOut; 
  char *tfile, *pfile, *ofile;
  
  int total_iterations;
  int pyramid_height; // number of iterations

/*	
  if (argc != 7) {
    usage(argc, argv);
  }

  if((grid_rows = atoi(argv[1]))<=0||
     (grid_cols = atoi(argv[1]))<=0||
     (pyramid_height = atoi(argv[2]))<=0||
     (total_iterations = atoi(argv[3]))<=0) {
    usage(argc, argv);
  }
		
  tfile=argv[4];
  pfile=argv[5];
  ofile=argv[6];
*/

  grid_rows = SIZE;
  grid_cols = SIZE; 

  total_iterations = ITER;
  pyramid_height = 2; 

#ifdef USEFILE
  tfile=argv[1];
  pfile=argv[2];
  ofile=argv[3];
#endif
	
  size=grid_rows*grid_cols;

  /* --------------- pyramid parameters --------------- */

  # define EXPAND_RATE 2 // add one iteration will extend the pyramid base by 2 per each borderline
  int borderCols = (pyramid_height)*EXPAND_RATE/2;
  int borderRows = (pyramid_height)*EXPAND_RATE/2;
  int smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
  int smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
  int blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0) ? 0:1);
  int blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0) ? 0:1);

  FilesavingTemp = (float *) malloc(size*sizeof(float));
  FilesavingPower = (float *) malloc(size*sizeof(float));
  MatrixOut = (float *) calloc (size, sizeof(float));

  if( !FilesavingPower || !FilesavingTemp || !MatrixOut) {
    fatal("unable to allocate memory");
  }

#ifdef VERBOSE
  printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",
	 pyramid_height, grid_cols, grid_rows, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);
#endif

#ifdef USEFILE
  readinput(FilesavingTemp, grid_rows, grid_cols, tfile);
  readinput(FilesavingPower, grid_rows, grid_cols, pfile);
#else
  generateinput(FilesavingPower, FilesavingTemp, grid_rows, grid_cols);
#endif

  float *MatrixTemp[2], *MatrixPower;
  hipMalloc((void**)&MatrixTemp[0], sizeof(float)*size);
  hipMalloc((void**)&MatrixTemp[1], sizeof(float)*size);
  hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);

  hipMalloc((void**)&MatrixPower, sizeof(float)*size);
  hipMemcpy(MatrixPower, FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);

#ifdef VERBOSE
  printf("Start computing the transient temperature\n");
#endif

  /* Main computation */
  int ret = compute_tran_temp( MatrixPower,MatrixTemp,grid_cols,grid_rows, 
                               total_iterations,pyramid_height, blockCols, 
                               blockRows, borderCols, borderRows);

#ifdef VERBOSE
  printf("Ending simulation\n");
#endif
  hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float)*size, hipMemcpyDeviceToHost);

#ifdef USEFILE
  writeoutput( MatrixOut,grid_rows, grid_cols, ofile);
#endif

  hipFree(MatrixPower);
  hipFree(MatrixTemp[0]);
  hipFree(MatrixTemp[1]);
  free(MatrixOut);
}